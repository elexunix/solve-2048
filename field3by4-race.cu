#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include "field3by4lib.h"

const long long arr_size = log_win_const * log_win_const * log_win_const * log_win_const * log_win_const * log_win_const
                    * (long long)log_win_const * log_win_const * log_win_const * log_win_const * log_win_const * log_win_const/* / 8*/ + 1;

// this causes race conditions, by the way
__host__ __device__ bool get_value(unsigned char *arr, long long pos) {
    return arr[pos];
    // return arr[pos >> 3] & 1 << (pos & 7);
}

__host__ __device__ void set_value(unsigned char *arr, long long pos, bool value) {
    arr[pos] = value;
    // if (value != get_value(arr, pos))
    //     arr[pos >> 3] ^= 1 << (pos & 7);
}

const long long grid_1dim_width = log_win_const * log_win_const * log_win_const * log_win_const * log_win_const * log_win_const;
const dim3 dimGrid(grid_1dim_width / 32 + 1, grid_1dim_width / 32 + 1), dimBlock(32, 32);

__global__ void process_layer(unsigned char *arr, int sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= grid_1dim_width || idy >= grid_1dim_width)
        return;
    long long id = idx * grid_1dim_width + idy;
    board position(decode_position(id));
    if (position.sum() != sum)
        return;
    bool winnable = false;
    for (int swipe = 0; swipe < 4; ++swipe) {
        board temp(position);
        bool possible;
        switch (swipe) {
            case 0: possible = temp.swipeLeft(); break;
            case 1: possible = temp.swipeRight(); break;
            case 2: possible = temp.swipeUp(); break;
            case 3: possible = temp.swipeDown(); break;
        }
        if (!possible)
            continue;
        if (temp.won())
            return set_value(arr, id, true);
        bool excellent = true;
        for (int shot = 0; shot < 24; ++shot) {
            board test(temp);
            if (test.addTile(shot))
                excellent &= get_value(arr, encode_position(test));
        }
        winnable |= excellent;
    }
    set_value(arr, id, winnable);
}

int main() {
    unsigned char *arr;
    report_start("allocating global array (" + itos(arr_size) + ")");
    gpuErrchk(hipMallocManaged(&arr, arr_size));
    report_finish();

    // report_start("filling global array with zeros");
    // fill_zeros<<<dimGrid, dimBlock>>>(arr);
    // hipDeviceSynchronize();
    // report_finish();

    for (int sum = 3 * 4 * win_const / 2; sum >= 0; sum -= 2) {
        report_start("processing layer with sum " + itos(sum));
        process_layer<<<dimGrid, dimBlock>>>(arr, sum);
        gpuErrchk(hipDeviceSynchronize());
        report_finish();
    }

    std::cout << "FINAL RESULT:\n";
    for (int i = 0; i < 3; ++i) {
        std::cout << '\t';
        for (int j = 0; j < 4; ++j) {
            board b;
            for (int k = 0; k < 3; ++k)
                for (int l = 0; l < 4; ++l)
                    b.f[k][l] = k == i && l == j ? 2 : 0;
            std::cout << get_value(arr, encode_position(b));
            for (int k = 0; k < 3; ++k)
                for (int l = 0; l < 4; ++l)
                    b.f[k][l] = k == i && l == j ? 4 : 0;
            std::cout << get_value(arr, encode_position(b)) << ' ';
        }
        std::cout << '\n';
    }

    // std::cout << "Discussed position (97734630):\n";
    // decode_position(97734630).print();
    /*while (true) {
        std::cout << "\n\nEnter position:\n";
        board b;
        for (int i = 0; i < 3; ++i) {
            std::cout << '\t';
            for (int j = 0; j < 4; ++j)
                std::cin >> b.f[i][j];
        }
        std::cout << "code: " << encode_position(b) << '\n';
        std::cout << "decoding for check:\n";
        decode_position(encode_position(b)).print();
        std::cout << "is won: " << b.won() << '\n';
        std::cout << "is lost: " << b.lost() << '\n';
        std::cout << "is finished: " << b.finished() << '\n';
        std::cout << "value: " << get_value(arr, encode_position(b)) << '\n';
    }*/
    reenter_pos:
    std::cout << "Enter position:\n";
    board b;
    for (int i = 0; i < 3; ++i) {
        std::cout << '\t';
        for (int j = 0; j < 4; ++j)
            std::cin >> b.f[i][j];
    }
    std::cout << "initial code: " << encode_position(b) << '\n';
    if (get_value(arr, encode_position(b))) {
        std::cout << "this position is winning... enter another one\n";
        goto reenter_pos;
    }
    assert(get_value(arr, encode_position(b)) == 0);
    while (true) {
        std::cout << "your swipe: ";
        int swipe;
        std::cin >> swipe;
        switch (swipe) {
            case 0: b.swipeLeft(); break;
            case 1: b.swipeRight(); break;
            case 2: b.swipeUp(); break;
            case 3: b.swipeDown(); break;
            case 4: goto reenter_pos;
        }
        std::cout << "position (" << encode_position(b) << ") after swipe:\n";
        b.print();
        for (int i = 0; i < 24; ++i) {
            board test(b);
            if (test.addTile(i) && !get_value(arr, encode_position(test))) {
                std::cout << "move " << i << " is the answer\n";
                b = test;
                break;
            }
        }
        std::cout << "now position (" << encode_position(b) << "):\n";
        b.print();
        std::cout << "its value: " << get_value(arr, encode_position(b)) << '\n';


    bool winnable = false;
    for (int swipe = 0; swipe < 4; ++swipe) {
        board temp(b);
        bool possible;
        switch (swipe) {
            case 0: possible = temp.swipeLeft(); break;
            case 1: possible = temp.swipeRight(); break;
            case 2: possible = temp.swipeUp(); break;
            case 3: possible = temp.swipeDown(); break;
        }
        if (!possible)
            continue;
        if (temp.won()) {
            std::cout << "out position " << encode_position(b) << " is almost won\n";
            continue;
        }
        bool excellent = true;
        for (int shot = 0; shot < 24; ++shot) {
            board test(temp);
            if (test.addTile(shot))
                excellent &= get_value(arr, encode_position(test));
        }
        winnable |= excellent;
    }
    std::cout << "winnable according to the rule: " << winnable << std::endl;
    }

    report_start("freeing memory");
    hipFree(arr);
    report_finish();
    return 0;
}